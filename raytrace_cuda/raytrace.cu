#include "hip/hip_runtime.h"
// cuda try implementation - Tomasz Michalik 05/22/18
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdbool.h>
#include <sys/time.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#define RADIUS 6.0
#define winMax 10

#define winY 10
#define PI 3.14159265359

typedef struct
{
    double x;
    double y;
    double z;
} vRay;


// 3D dot product
__device__
double dotproduct(vRay *V1, vRay *V2)
{
    return V1->x*V2->x + V1->y*V2->y + V1->z*V2->z;
}



// view
__device__
bool view_check(vRay * V, vRay * C, double * t)
{
    double VC = dotproduct(V, C);
    double CC = dotproduct(C, C);

    if ((VC*VC + RADIUS*RADIUS - CC) < 0)
        return false;

    *t = VC - sqrt(VC*VC +RADIUS*RADIUS - CC);

    return true;
}


// intersection of view and sphere
__device__
void intersect(vRay * I, vRay * V, double t)
{
    I->x = t*V->x;

    I->y = t*V->y;
    I->z = t*V->z;
}


// unit normal vector
__device__
void ray_normal(vRay *I, vRay *C, vRay *N)
{
    vRay ImC;

    ImC.x = I->x - C->x;
    ImC.y = I->y - C->y;
    ImC.z = I->z - C->z;

    double dot_ImC = dotproduct(&ImC, &ImC);

    N->x = (ImC.x) / sqrt(dot_ImC);
    N->y = (ImC.y) / sqrt(dot_ImC);
    N->z = (ImC.z) / sqrt(dot_ImC);
}


// create shadow ray and compute brightness
__device__
double brightness(vRay *I, vRay *L, vRay *N)
{
    vRay S;
    vRay LmI;

    LmI.x = L->x - I->x;
    LmI.y = L->y - I->y;
    LmI.z = L->z - I->z;

    double dot_LmI = dotproduct(&LmI, &LmI);

    S.x = (LmI.x) / sqrt(dot_LmI);
    S.y = (LmI.y) / sqrt(dot_LmI);
    S.z = (LmI.z) / sqrt(dot_LmI);

    // return the max between 0 and S.N
    double dots = dotproduct(&S, N);

    return (dots > 0) ? dots : 0.0;
}


// kernel function
__global__
void sampAlgo(double *grid, int n, int n_rays)
{
    // use cuda's random number generator
    int i =  blockDim.x*blockIdx.x + threadIdx.x;

    hiprandState_t state;
    hiprand_init(i, 0, 0, &state);

    double delta = (((double)n) / ((double)(2*winMax)));

    // set up light source and sphere center position
    vRay L;
    L.x = 4; L.y = 4; L.z = -1;

    vRay C;
    C.x = 0; C.y = 12; C.z = 0;

    vRay W; vRay V; vRay I; vRay N;

    for (int i = 0; i < n; ++i)
    {
        double t, theta, phi, b;

        do
        {
            phi = (double) hiprand_uniform(&state) * (double) M_PI;
            theta = (double) hiprand_uniform(&state)  * (double) M_PI;

            V.x  = sin(theta) * cos(phi);
            V.y  = sin(theta) * sin(phi);
            V.z  = cos(theta);

            W.x = (winY / V.y) * V.x;
            W.y = (winY / V.y) * V.y;
            W.z = (winY / V.y) * V.z;

        } while ((!view_check(&V, &C,  &t)) || (fabs(W.x) > winMax) || (fabs(W.z) > winMax));
// change to && , || produces similar timings
        intersect(&I, &V, t);

        ray_normal(&I, &C, &N);

        b = brightness(&I, &L, &N);

        double x = (W.x + (double)winMax);
        double z = (W.z + (double)winMax);
        x = x * delta;
        z = z * delta;

        grid[(int)x*n + (int)z] += b;

    }
}



int main(int argc, char **argv)
{
    // arg[1] = number of rays, arg[2] = number of grid points
    if (argc != 3)
    {
        printf("Invalid number of arguments.\n");
        exit(1);
    }

    int n = atoi(argv[1]);
    int n_rays = atoi(argv[2]);
    struct timeval start, end;
    gettimeofday(&start, NULL);

    srand(time(NULL));

    // allocate window (n x n)
    double * grid = (double *) calloc(n*n, sizeof(double));

    // copy data over launch kernel

    // Cuda malloc
    double * cuda_grid;
    hipError_t _e;
    _e = hipMalloc((void**)&cuda_grid, n*n * sizeof(double));
    if (_e != hipSuccess)
        printf("Report Cuda error: %s\n", hipGetErrorString(_e));

    //transfer  to gpu
    _e = hipMemcpy(cuda_grid, grid, n*n*sizeof(double), hipMemcpyHostToDevice);
    if (_e != hipSuccess)
        printf("Report Cuda error: %s\n", hipGetErrorString(_e));

    // run kernel
    int block_size =; // modify here
    int rays_per_thread = 10; // set default value
    int n_blocks = (n_rays + block_size - 1) / (block_size*rays_per_thread);
    printf("threads per block:%d number of blocks = %d\n", block_size, n_blocks);
    // blocks, threads per block parameters setx
    sampAlgo<<< n_blocks, block_size>>>(cuda_grid, n, rays_per_thread);
    _e = hipGetLastError();

    //get from gpu
    _e = hipMemcpy(grid, cuda_grid, n*n*sizeof(double), hipMemcpyDeviceToHost);
    if (_e != hipSuccess)
        printf("Cuda error: %s\n", hipGetErrorString(_e));

    gettimeofday(&end, NULL);
    double m = 1000000;
    double t = ((end.tv_sec*m + end.tv_usec) - (start.tv_sec*m + start.tv_usec));
    printf("Cuda,%d, %g\n", n_rays, t / m);

    // write grid to file
    FILE * file = fopen("rays.out", "wb");

    for (int i = 0; i < n; ++i)
        for (int j = 0; j < n; ++j)
        {
            fwrite(&(grid[i*n + j]), sizeof(double), 1, file);
        }


    fclose(file);

    free(grid);

    return 0;
}